
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>


void MVM(double* C, const double* A, const double* B, long N) {
  for (long i = 0; i < N; i++) {
    double sum = 0;
    #pragma omp parallel for schedule(static) reduction(+:sum)
    for (long j = 0; j < N; j++) {
      sum += A[i*N+j]*B[j];
    }
    C[i] = sum;
  }
}

#define BLOCK_SIZE 1024

__global__ void reduction(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void product(double* sum, const double* A, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = A[idx]*b[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

int main() {
  long N = (1UL<<10);
  double *x, *M;

  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&M, N*N*sizeof(double), hipHostMallocDefault);

  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = drand48();
  }
  for (long i = 0; i < N*N; i++) {
    M[i] = drand48();
  }

  double *sum_ref, *sum;
  hipHostMalloc((void**)&sum_ref, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&sum, N * sizeof(double), hipHostMallocDefault);

  double tt = omp_get_wtime();
  MVM(sum_ref, M, x, N);
  printf("CPU Bandwidth = %f GB/s\n", N*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("CPU test %f \n", sum_ref[1]);

  double *x_d, *M_d, *z_d;
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&M_d, N*N*sizeof(double));
  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(M_d, M, N*N*sizeof(double), hipMemcpyHostToDevice);
    
  long u = 1;
  for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)){
    u += i;
  }
  hipMalloc(&z_d, u*sizeof(double)); 


  hipDeviceSynchronize();

  tt = omp_get_wtime();

  for (long i = 0; i < N; i++) {
    double* sum_d = z_d;
    long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
    product<<<Nb,BLOCK_SIZE>>>(sum_d, M_d+i*N, x_d, N);
    while (Nb > 1) {
      long Nx = Nb;
      Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
      reduction<<<Nb,BLOCK_SIZE>>>(sum_d + Nx, sum_d, Nx);
      sum_d += Nx;
    }
    hipMemcpyAsync(&sum[i], sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }

  printf("GPU Bandwidth = %f GB/s\n", N*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double error = 0;
  #pragma omp parallel for reduction(+:error)
  for (long i = 0; i < N; i++) {
    error = error + fabs(sum[i] - sum_ref[i]);
  }
  printf("Error = %f\n", error);
  
  hipFree(x_d);
  hipFree(z_d);
  hipFree(M_d); 
  hipHostFree(x);
  hipHostFree(M);

  return 0;
}